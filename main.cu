#include "data_loader.h"
#include "photo_mosaic_serial.h"
#include "photo_mosaic_cuda.cuh"

int main(){

	Photo_Mosaic_Serial *photo_mosaic_serial;
	Photo_Mosaic_Cuda *photo_mosaic_cuda;
	RGBImage *result_serial, *result_cuda;

	// test case1: owl
	photo_mosaic_serial = new Photo_Mosaic_Serial();
	result_serial = photo_mosaic_serial->Run("Image-Folder/4k_owl.jpg","Image-Folder/cifar10");
	result_serial->DumpImage("4k_owl_serial.jpg");
	result_serial->Display_CMD();
	photo_mosaic_cuda = new Photo_Mosaic_Cuda();
	result_cuda = photo_mosaic_cuda->Run("Image-Folder/4k_owl.jpg","Image-Folder/cifar10");
	result_cuda->DumpImage("4k_owl_cuda.jpg");
	result_cuda->Display_CMD();
	delete result_serial;
	delete result_cuda;
	delete photo_mosaic_serial;
	delete photo_mosaic_cuda;

	// test case2: sunflower
	photo_mosaic_serial = new Photo_Mosaic_Serial();
	result_serial = photo_mosaic_serial->Run("Image-Folder/sunflower.jpg","Image-Folder/cifar10");
	result_serial->DumpImage("sunflower_serial.jpg");
	result_serial->Display_CMD();
	photo_mosaic_cuda = new Photo_Mosaic_Cuda();
	result_cuda = photo_mosaic_cuda->Run("Image-Folder/sunflower.jpg","Image-Folder/cifar10");
	result_cuda->DumpImage("sunflower_cuda.jpg");
	result_cuda->Display_CMD();
	delete result_serial;
	delete result_cuda;
	delete photo_mosaic_serial;
	delete photo_mosaic_cuda;

	// test case3: dogs
	photo_mosaic_serial = new Photo_Mosaic_Serial();
	result_serial = photo_mosaic_serial->Run("Image-Folder/dog.jpg","Image-Folder/cifar10");
	result_serial->DumpImage("dog_serial.jpg");
	result_serial->Display_CMD();
	photo_mosaic_cuda = new Photo_Mosaic_Cuda();
	result_cuda = photo_mosaic_cuda->Run("Image-Folder/dog.jpg","Image-Folder/cifar10");
	result_cuda->DumpImage("dog_cuda.jpg");
	result_cuda->Display_CMD();
	delete result_serial;
	delete result_cuda;
	delete photo_mosaic_serial;
	delete photo_mosaic_cuda;

	return 0;
}
