
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

int main(){
	int *a, *b, *c;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;	// device copies of a, b, c
	int size = N * sizeof(int);

	// allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// allocate space for host copies of a, b, c
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for(int i = 0; i < N; i++){
		a[i] = i;
		b[i] = i;
	}

	// copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// launch add kernel on GPU with N threads
	add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

	// copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++){
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	// cleanup
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
