
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <stdio.h>

// both a, b, c are the pointer on the device memory
// should allocate memory on the device first!
__global__ void add(int *a, int *b, int *c){
	*c = *a + *b;
}

int main(){
	int a, b, c;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;	// device copies of a, b, c
	int size = sizeof(int);

	// allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// setup input values
	a = 2;
	b = 7;

	// copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// launch add kernel on GPU
	add<<<1, 1>>>(d_a, d_b, d_c);

	// copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	// cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("c = %d\n", c);
	return 0;
}
