
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512

__global__ void add(int *a, int *b, int *c){
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(){
	int *a, *b, *c;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;	// device copies of a, b, c
	int size = N * sizeof(int);

	// allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// allocate space for host copies of a, b, c
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for(int i = 0; i < N; i++){
		a[i] = i;
		b[i] = i;
	}

	// copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// launch add kernel on GPU with N threads
	add<<<1, N>>>(d_a, d_b, d_c);

	// copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++){
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	// cleanup
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
